#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

#include <cstdio>
#include <cassert>


#define check_cuda_call(ans) { _check((ans), __FILE__, __LINE__); }
inline void _check(hipError_t code, char *file, int line)
{
  if (code != hipSuccess) {
    fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);
  }
}


class EventTimer {
public:
  EventTimer() : mStarted(false), mStopped(false) {
    hipEventCreate(&mStart);
    hipEventCreate(&mStop);
  }
  ~EventTimer() {
    hipEventDestroy(mStart);
    hipEventDestroy(mStop);
  }
  void start(hipStream_t s = 0) {
    hipEventRecord(mStart, s); 
    mStarted = true;
    mStopped = false;
  }
  void stop(hipStream_t s = 0)  {
    assert(mStarted);
    hipEventRecord(mStop, s); 
    mStarted = false;
    mStopped = true;
  }
  float elapsed() {
    assert(mStopped);
    if (!mStopped) return 0; 
    hipEventSynchronize(mStop);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, mStart, mStop);
    return elapsed;
  }

private:
  bool mStarted, mStopped;
  hipEvent_t mStart, mStop;
};


__global__ void state_setup(hiprandStateXORWOW_t* states, int w, int h)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  if (x >= w || y >= h) {
    return;
  }
  int i = x + y * w;
  hiprand_init(clock64(), x, 0, states + i);
}


__global__ void write(int *buf, int w, int h, float k, hiprandStateXORWOW_t* states)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  if (x >= w || y >= h) {
    return;
  }
  int i = x + y * w;
  buf[i] = hiprand(states + i) & 1 ? 0x00000000 : 0xffffffff;
  // (x << 24 | y << 16 | x << 8 | y) * k;
}


hiprandStateXORWOW_t* states;

void state_setup(int w, int h)
{
  check_cuda_call(hipMalloc(&states, w * h * sizeof(hiprandStateXORWOW_t)));
  check_cuda_call(hipMemset(states, 0, w * h * sizeof(hiprandStateXORWOW_t)));
  dim3 dim_block(32, 16); // 32 * 16 = 512;
  dim3 dim_grid(((w + dim_block.x - 1) / dim_block.x),
                 (h + dim_block.y - 1) / dim_block.y);
  state_setup<<<dim_grid, dim_block>>>(states, w, h);
}


void cuda_write(int* buf, int w, int h, float k)
{
  dim3 dim_block(32, 16); // 32 * 16 = 512;
  dim3 dim_grid(((w + dim_block.x - 1) / dim_block.x),
                 (h + dim_block.y - 1) / dim_block.y);
  EventTimer t;
  t.start();
  write<<<dim_grid, dim_block>>>(buf, w, h, k, states);
  t.stop();
  //printf("kernel time: %f\n", t.elapsed());
}


void state_destroy()
{
  check_cuda_call(hipFree(states));
}








